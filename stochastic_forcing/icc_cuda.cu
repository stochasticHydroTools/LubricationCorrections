#include "hip/hip_runtime.h"
#include "icc_cuda.h"
#include <fstream>
#include <thrust/version.h>
// #include <thrust/reduce.h>
// #include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
// #include <thrust/transform_reduce.h>
// #include <thrust/functional.h>
#include <thrust/sort.h>
// #include </usr/include/python2.6/Python.h>

#define chkErrq(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess) 
  {
    cout << "GPUasser: " << hipGetErrorString(code) << "   "  << file << "  "  << line << endl;
    if (abort) exit(code);
  }
}

#define chkErrqCusparse(ans) { cusparseAssert((ans), __FILE__, __LINE__); }
inline void cusparseAssert(hipsparseStatus_t code, const char *file, int line, bool abort=true)
{
  if (code != 0) 
  {
    if(code == 1)
      cout << code << " cusparseStatusNotInitialized " << file << "  " << line << endl;
    else if(code == 2)
      cout << code << " cusparseStatusAllocFailed " << file << "  " << line << endl;
    else if (code == 3)
      cout << code << " cusparseStatusInvalidValue " << file << "  " << line << endl;
    else if (code == 4)
      cout << code << " cusparseStatusArchMismatch " << file << "  " << line << endl;
    else if (code == 5)
      cout << code << " cusparseStatusMappingError " << file << "  " << line << endl;
    else if (code == 6)
      cout << code << " cusparseStatusExecutionFailed " << file << "  " << line << endl;
    else if (code == 7)
      cout << code << " cusparseStatusInternalError " << file << "  " << line << endl;
    else if (code == 8)
      cout << code << " cusparseStatusMatrixTypeNotSupported " << file << "  " << line << endl;
    // cout << "cuSparseasser: " << code << "   "  << file << "  "  << line << endl;
    if (abort) exit(code);
  }
}

struct saxpy_functor
{
  const int m;
  saxpy_functor(int _m) : m(_m) {}

    __host__ __device__
    unsigned long long int operator()(const int& x, const unsigned long long int& y) const { 
      return (unsigned long long) m * (unsigned long long) x + y;
    }
};

void saxpy_fast(int m, thrust::device_vector<int>& X, thrust::device_vector<unsigned long long int>& Y)
{
  // Y <- m * X + Y
  thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(m));
}

int print_csr_matrix_in_dense_format(hipsparseHandle_t handle, 
				     int num_rows, 
				     int num_col, 
				     int nnz,
				     const hipsparseMatDescr_t descr,
				     const double *csrVal, 
				     const int *csrRowPtr,
				     const int *csrColInd,
				     const double *x){
  int size = num_rows * num_col;
  // Allocate memory
  double *A = new double [size];
  double *A_gpu;
  chkErrq(hipMalloc((void**)&A_gpu, size * sizeof(double)));
  // Copy matrix to dense format and print
  hipsparseMatrixType_t mat_type = hipsparseGetMatType(descr);
  chkErrqCusparse(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  hipsparseDcsr2dense(handle, num_rows, num_col, descr, csrVal, csrRowPtr, csrColInd, A_gpu, num_rows);
  chkErrqCusparse(hipsparseSetMatType(descr, mat_type));
  chkErrq(hipMemcpy(A, A_gpu, size * sizeof(double), hipMemcpyDeviceToHost));  
  printf("Matrix = \n");
  for(int i=0; i<num_rows; i++){
    for(int j=0; j<num_col; j++){
      printf("%010f  ", A[i*num_col + j]);
    }
    printf("\n");
  }
  printf("\n\n");
  // Free memory
  chkErrq(hipFree(A_gpu));
  delete[] A;  
  return 0;
}
				      

/*
 mobilityUFRPY computes the 3x3 RPY mobility
 between blobs i and j normalized with 8 pi eta a
*/
__device__ void mobilityUFRPY(double rx,
			      double ry,
			      double rz,
			      double &Mxx,
			      double &Mxy,
			      double &Mxz,
			      double &Myy,
			      double &Myz,
			      double &Mzz,
			      int i,
			      int j,
                              double invaGPU){
  
  double fourOverThree = 4.0 / 3.0;

  if(i==j){
    Mxx = fourOverThree;
    Mxy = 0;
    Mxz = 0;
    Myy = Mxx;
    Myz = 0;
    Mzz = Mxx;
  }
  else{
    rx = rx * invaGPU; //Normalize distance with hydrodynamic radius
    ry = ry * invaGPU;
    rz = rz * invaGPU;
    double r2 = rx*rx + ry*ry + rz*rz;
    double r = sqrt(r2);
    //We should not divide by zero but std::numeric_limits<double>::min() does not work in the GPU
    //double invr = (r > std::numeric_limits<double>::min()) ? (1.0 / r) : (1.0 / std::numeric_limits<double>::min())
    double invr = 1.0 / r;
    double invr2 = invr * invr;
    double c1, c2;
    if(r>=2){
      c1 = 1 + 2 / (3 * r2);
      c2 = (1 - 2 * invr2) * invr2;
      Mxx = (c1 + c2*rx*rx) * invr;
      Mxy = (     c2*rx*ry) * invr;
      Mxz = (     c2*rx*rz) * invr;
      Myy = (c1 + c2*ry*ry) * invr;
      Myz = (     c2*ry*rz) * invr;
      Mzz = (c1 + c2*rz*rz) * invr;
    }
    else{
      c1 = fourOverThree * (1 - 0.28125 * r); // 9/32 = 0.28125
      c2 = fourOverThree * 0.09375 * invr;    // 3/32 = 0.09375
      Mxx = c1 + c2 * rx*rx ;
      Mxy =      c2 * rx*ry ;
      Mxz =      c2 * rx*rz ;
      Myy = c1 + c2 * ry*ry ;
      Myz =      c2 * ry*rz ;
      Mzz = c1 + c2 * rz*rz ;
    }
  } 
  return;
}


/*
 mobilityRPY computes the 3x3 mobility correction due to a wall
 between blobs i and j normalized with 8 pi eta a.
 This uses the expression from the Swan and Brady paper for a finite size particle.
 Mobility is normalize by 8*pi*eta*a.
*/
__device__ void mobilityUFSingleWallCorrection(double rx,
			                       double ry,
			                       double rz,
			                       double &Mxx,
                  			       double &Mxy,
			                       double &Mxz,
                                               double &Myx,
			                       double &Myy,
			                       double &Myz,
                                               double &Mzx,
                                               double &Mzy,
			                       double &Mzz,
			                       int i,
			                       int j,
                                               double invaGPU,
                                               double hj){
  if(i == j){
    double invZi = 1.0 / hj;
    Mxx += -(9*invZi - 2*pow(invZi,3) + pow(invZi,5)) / 12.0;
    Myy += -(9*invZi - 2*pow(invZi,3) + pow(invZi,5)) / 12.0;
    Mzz += -(9*invZi - 4*pow(invZi,3) + pow(invZi,5)) / 6.0;
  }
  else{
    double h_hat = hj / rz;
    double invR = rsqrt(rx*rx + ry*ry + rz*rz); // = 1 / r;
    double ex = rx * invR;
    double ey = ry * invR;
    double ez = rz * invR;
    
    double fact1 = -(3*(1+2*h_hat*(1-h_hat)*ez*ez) * invR + 2*(1-3*ez*ez) * pow(invR,3) - 2*(1-5*ez*ez) * pow(invR,5))  / 3.0;
    double fact2 = -(3*(1-6*h_hat*(1-h_hat)*ez*ez) * invR - 6*(1-5*ez*ez) * pow(invR,3) + 10*(1-7*ez*ez) * pow(invR,5)) / 3.0;
    double fact3 =  ez * (3*h_hat*(1-6*(1-h_hat)*ez*ez) * invR - 6*(1-5*ez*ez) * pow(invR,3) + 10*(2-7*ez*ez) * pow(invR,5)) * 2.0 / 3.0;
    double fact4 =  ez * (3*h_hat*invR - 10*pow(invR,5)) * 2.0 / 3.0;
    double fact5 = -(3*h_hat*h_hat*ez*ez*invR + 3*ez*ez*pow(invR, 3) + (2-15*ez*ez)*pow(invR, 5)) * 4.0 / 3.0;
    
    Mxx += fact1 + fact2 * ex*ex;
    Mxy += fact2 * ex*ey;
    Mxz += fact2 * ex*ez + fact3 * ex;
    Myx += fact2 * ey*ex;
    Myy += fact1 + fact2 * ey*ey;
    Myz += fact2 * ey*ez + fact3 * ey;
    Mzx += fact2 * ez*ex + fact4 * ex;
    Mzy += fact2 * ez*ey + fact4 * ey;
    Mzz += fact1 + fact2 * ez*ez + fact3 * ez + fact4 * ez + fact5;         
  }
}


/*
 velocity_from_force computes the product
 U = M*F
*/
__global__ void velocity_from_force(const double *x,
                                    const double *f,					
                                    double *u,
				    int number_of_blobs,
                                    double eta,
                                    double a){
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i >= number_of_blobs) return;   

  double invaGPU = 1.0 / a;

  double Ux=0;
  double Uy=0;
  double Uz=0;

  double rx, ry, rz;

  double Mxx, Mxy, Mxz;
  double Myx, Myy, Myz;
  double Mzx, Mzy, Mzz;

  int NDIM = 3; // 3 is the spatial dimension
  int ioffset = i * NDIM; 
  int joffset;
  
  for(int j=0; j<number_of_blobs; j++){
    joffset = j * NDIM;

    // Compute vector between particles i and j
    rx = x[ioffset    ] - x[joffset    ];
    ry = x[ioffset + 1] - x[joffset + 1];
    rz = x[ioffset + 2] - x[joffset + 2];

    // 1. Compute mobility for pair i-j
    mobilityUFRPY(rx,ry,rz, Mxx,Mxy,Mxz,Myy,Myz,Mzz, i,j, invaGPU);
    Myx = Mxy;
    Mzx = Mxz;
    Mzy = Myz;
    mobilityUFSingleWallCorrection(rx/a, ry/a, (rz+2*x[joffset+2])/a, Mxx,Mxy,Mxz,Myx,Myy,Myz,Mzx,Mzy,Mzz, i,j, invaGPU, x[joffset+2]/a);

    //2. Compute product M_ij * F_j
    Ux = Ux + (Mxx * f[joffset] + Mxy * f[joffset + 1] + Mxz * f[joffset + 2]);
    Uy = Uy + (Myx * f[joffset] + Myy * f[joffset + 1] + Myz * f[joffset + 2]);
    Uz = Uz + (Mzx * f[joffset] + Mzy * f[joffset + 1] + Mzz * f[joffset + 2]);
  }
  //LOOP END

  //3. Save velocity U_i
  double pi = 4.0 * atan(1.0);
  double norm_fact_f = 8 * pi * eta * a;
  u[ioffset    ] = Ux / norm_fact_f;
  u[ioffset + 1] = Uy / norm_fact_f;
  u[ioffset + 2] = Uz / norm_fact_f;

  return;
}


/*
  Determine number of non-zero elements (nnz)
*/
__global__ void countNnz(const double *x, unsigned long long int *nnzGPU, const double cutoff, const int N){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i >= N) return;   

  double rx, ry, rz, r2;
  int NDIM = 3; // 3 is the spatial dimension
  int ioffset = i * NDIM; 
  int joffset;
  
  // Loop over columns
  for(int j=0; j<N; j++){
    joffset = j * NDIM;
    
    // Compute vector between blobs i and j
    rx = x[ioffset    ] - x[joffset    ];
    ry = x[ioffset + 1] - x[joffset + 1];
    rz = x[ioffset + 2] - x[joffset + 2];
    r2 = (rx*rx + ry*ry + rz*rz);
    
    // If blobs are close increse nnz
    if(r2 < cutoff*cutoff){
      unsigned long long int nnz_old = atomicAdd(nnzGPU, 9);
    }
  }
}


/*
  Build a sparse lower triangular matrix with coordinated format (COO). See cuSparse documentation.
*/
__global__ void buildLowerTriangularCOOMatrix(const double *x,
			                      double *cooValA,
                                              int *cooRowIndA,
                                              int *cooColIndA,
                                              unsigned long long int *nnzGPU,
			                      const double eta,
			                      const double a,
			                      const double cutoff,
			                      const int N){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i >= N) return;   

  double pi = 4.0 * atan(1.0);
  double norm_fact = 1.0 / (8 * pi * eta * a);  
  double inva = 1.0 / a;
  double rx, ry, rz, r2;
  int NDIM = 3; // 3 is the spatial dimension
  int ioffset = i * NDIM; 
  int joffset;
  double Mxx, Mxy, Mxz;
  double Myx, Myy, Myz;
  double Mzx, Mzy, Mzz;

  // Loop over columns
  for(int j=i; j<N; j++){
    joffset = j * NDIM;
    
    // Compute vector between blobs i and j
    rx = x[ioffset    ] - x[joffset    ];
    ry = x[ioffset + 1] - x[joffset + 1];
    rz = x[ioffset + 2] - x[joffset + 2];
    r2 = (rx*rx + ry*ry + rz*rz);
    
    // If blobs are close compute pair-mobility
    if(r2 < cutoff*cutoff){
      mobilityUFRPY(rx,ry,rz, Mxx,Mxy,Mxz,Myy,Myz,Mzz, i,j, inva);
      Myx = Mxy;
      Mzx = Mxz;
      Mzy = Myz;
      mobilityUFSingleWallCorrection(rx/a, ry/a, (rz+2*x[joffset+2])/a, Mxx,Mxy,Mxz,Myx,Myy,Myz,Mzx,Mzy,Mzz, i,j, inva, x[joffset+2]/a);
      
      if( i != j){
        int nnz_old = atomicAdd(nnzGPU, 9);      
        cooValA[nnz_old] = Mxx * norm_fact;
        cooRowIndA[nnz_old] = ioffset;
        cooColIndA[nnz_old] = joffset;

        nnz_old++;
        cooValA[nnz_old] = Mxy * norm_fact;
        cooRowIndA[nnz_old] = ioffset;
        cooColIndA[nnz_old] = joffset + 1;

        nnz_old++;
        cooValA[nnz_old] = Mxz * norm_fact;
        cooRowIndA[nnz_old] = ioffset;
        cooColIndA[nnz_old] = joffset + 2;

        nnz_old++;
        cooValA[nnz_old] = Myx * norm_fact;
        cooRowIndA[nnz_old] = ioffset + 1;
        cooColIndA[nnz_old] = joffset;

        nnz_old++;
        cooValA[nnz_old] = Myy * norm_fact;
        cooRowIndA[nnz_old] = ioffset + 1;
        cooColIndA[nnz_old] = joffset + 1;

        nnz_old++;
        cooValA[nnz_old] = Myz * norm_fact;
        cooRowIndA[nnz_old] = ioffset + 1;
        cooColIndA[nnz_old] = joffset + 2;

        nnz_old++;
        cooValA[nnz_old] = Mzx * norm_fact;
        cooRowIndA[nnz_old] = ioffset + 2;
        cooColIndA[nnz_old] = joffset ;

        nnz_old++;
        cooValA[nnz_old] = Mzy * norm_fact;
        cooRowIndA[nnz_old] = ioffset + 2;
        cooColIndA[nnz_old] = joffset + 1;

        nnz_old++;
        cooValA[nnz_old] = Mzz * norm_fact;
        cooRowIndA[nnz_old] = ioffset + 2;
        cooColIndA[nnz_old] = joffset + 2;
      }
      else{
        int nnz_old = atomicAdd(nnzGPU, 6); //
        cooValA[nnz_old] = Mxx * norm_fact;
        cooRowIndA[nnz_old] = ioffset;
        cooColIndA[nnz_old] = joffset;

	nnz_old++; //
	cooValA[nnz_old] = Mxy * norm_fact; //
	cooRowIndA[nnz_old] = ioffset;      //
	cooColIndA[nnz_old] = joffset + 1;  //

	nnz_old++; //
	cooValA[nnz_old] = Mxz * norm_fact; //
	cooRowIndA[nnz_old] = ioffset; //
	cooColIndA[nnz_old] = joffset + 2; //

        nnz_old++;
        cooValA[nnz_old] = Myy * norm_fact;
        cooRowIndA[nnz_old] = ioffset + 1;
        cooColIndA[nnz_old] = joffset + 1;

	nnz_old++; //
	cooValA[nnz_old] = Myz * norm_fact; //
	cooRowIndA[nnz_old] = ioffset + 1; //
	cooColIndA[nnz_old] = joffset + 2; //

        nnz_old++;
        cooValA[nnz_old] = Mzz * norm_fact;
        cooRowIndA[nnz_old] = ioffset + 2;
        cooColIndA[nnz_old] = joffset + 2;
      }
    } 
  }  
} 


/*
  Determine number of non-zero elements (nnz) in a lower
  Triangular matrix.
*/
__global__ void countLowerTriangularNnz(const double *x, unsigned long long int *nnzGPU, const double cutoff, const int N){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i >= N) return;   

  double rx, ry, rz, r2;
  int NDIM = 3; // 3 is the spatial dimension
  int ioffset = i * NDIM; 
  int joffset;
  
  // Loop over columns
  for(int j=i; j<N; j++){
    joffset = j * NDIM;
    
    // Compute vector between blobs i and j
    rx = x[ioffset    ] - x[joffset    ];
    ry = x[ioffset + 1] - x[joffset + 1];
    rz = x[ioffset + 2] - x[joffset + 2];
    r2 = (rx*rx + ry*ry + rz*rz);
    
    // If blobs are close increse nnz
    if(r2 < cutoff*cutoff){
      if( i != j){
        atomicAdd(nnzGPU, 9);
      }
      else{
        atomicAdd(nnzGPU, 6); //
      }
    }
  }
}



/*
  Build a sparse matrix with coordinated format (COO). See cuSparse documentation.
*/
__global__ void buildCOOMatrix(const double *x,
			       double *cooValA,
                               int *cooRowIndA,
                               int *cooColIndA,
                               unsigned long long int *nnzGPU,
			       const double eta,
			       const double a,
			       const double cutoff,
			       const int N){

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if(i >= N) return;   

  double pi = 4.0 * atan(1.0);
  double norm_fact = 1.0 / (8 * pi * eta * a);  
  double inva = 1.0 / a;
  double rx, ry, rz, r2;
  int NDIM = 3; // 3 is the spatial dimension
  int ioffset = i * NDIM; 
  int joffset;
  double Mxx, Mxy, Mxz;
  double Myx, Myy, Myz;
  double Mzx, Mzy, Mzz;

  // Loop over columns
  for(int j=0; j<N; j++){
    joffset = j * NDIM;
    
    // Compute vector between blobs i and j
    rx = x[ioffset    ] - x[joffset    ];
    ry = x[ioffset + 1] - x[joffset + 1];
    rz = x[ioffset + 2] - x[joffset + 2];
    r2 = (rx*rx + ry*ry + rz*rz);
    
    // If blobs are close compute pair-mobility
    if(r2 < cutoff*cutoff){
      mobilityUFRPY(rx,ry,rz, Mxx,Mxy,Mxz,Myy,Myz,Mzz, i,j, inva);
      Myx = Mxy;
      Mzx = Mxz;
      Mzy = Myz;
      mobilityUFSingleWallCorrection(rx/a, ry/a, (rz+2*x[joffset+2])/a, Mxx,Mxy,Mxz,Myx,Myy,Myz,Mzx,Mzy,Mzz, i,j, inva, x[joffset+2]/a);
      
      int nnz_old = atomicAdd(nnzGPU, 9);      
      cooValA[nnz_old] = Mxx * norm_fact;
      cooRowIndA[nnz_old] = ioffset;
      cooColIndA[nnz_old] = joffset;

      nnz_old++;
      cooValA[nnz_old] = Mxy * norm_fact;
      cooRowIndA[nnz_old] = ioffset;
      cooColIndA[nnz_old] = joffset + 1;

      nnz_old++;
      cooValA[nnz_old] = Mxz * norm_fact;
      cooRowIndA[nnz_old] = ioffset;
      cooColIndA[nnz_old] = joffset + 2;

      nnz_old++;
      cooValA[nnz_old] = Myx * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 1;
      cooColIndA[nnz_old] = joffset;

      nnz_old++;
      cooValA[nnz_old] = Myy * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 1;
      cooColIndA[nnz_old] = joffset + 1;

      nnz_old++;
      cooValA[nnz_old] = Myz * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 1;
      cooColIndA[nnz_old] = joffset + 2;

      nnz_old++;
      cooValA[nnz_old] = Mzx * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 2;
      cooColIndA[nnz_old] = joffset ;

      nnz_old++;
      cooValA[nnz_old] = Mzy * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 2;
      cooColIndA[nnz_old] = joffset + 1;

      nnz_old++;
      cooValA[nnz_old] = Mzz * norm_fact;
      cooRowIndA[nnz_old] = ioffset + 2;
      cooColIndA[nnz_old] = joffset + 2;
    } 
  }  
} 


/*
  Constructor: build the sparse mobility matrix M
  and compute the Cholesky factorization M=L*L.T
  where L is a lower triangular matrix.
*/
icc::icc(const double blob_radius, 
	 const double eta, 
	 const double cutoff,
	 const int number_of_blobs,
	 const double *x){
  d_icc_is_initialized = 0;
  d_blob_radius = blob_radius;
  d_eta = eta;
  d_cutoff = cutoff;
  d_number_of_blobs = number_of_blobs;
  d_x = x;

  // Determine number of blocks and threads for the GPU
  d_threads_per_block = 512;
  if((d_number_of_blobs / d_threads_per_block) < 512){
    d_threads_per_block = 256;
  }
  if((d_number_of_blobs / d_threads_per_block) < 256){
    d_threads_per_block = 128;
  }
  if((d_number_of_blobs / d_threads_per_block) < 128){
    d_threads_per_block = 128;
  }
  if((d_number_of_blobs / d_threads_per_block) < 128){
    d_threads_per_block = 64;
  }
  if((d_number_of_blobs / d_threads_per_block) < 32){
    d_threads_per_block = 128;
  }
  d_num_blocks = (d_number_of_blobs - 1) / d_threads_per_block + 1;
}


/*
  Constructor: build the sparse mobility matrix M
  and compute the Cholesky factorization M=L*L.T
  where L is a lower triangular matrix.
*/
icc::icc(const double blob_radius, 
	 const double eta, 
	 const double cutoff,
	 const int number_of_blobs,
	 bp::object x_obj){
  d_icc_is_initialized = 0;
  d_blob_radius = blob_radius;
  d_eta = eta;
  d_cutoff = cutoff;
  d_number_of_blobs = number_of_blobs;

  // Extract pointer
  PyObject* pobj = x_obj.ptr();
  Py_buffer pybuf;
  PyObject_GetBuffer(pobj, &pybuf, PyBUF_SIMPLE);
  void *buf = pybuf.buf;
  double *x = (double*)buf;
  d_x = x;

  // Determine number of blocks and threads for the GPU
  d_threads_per_block = 512;
  if((d_number_of_blobs / d_threads_per_block) < 512){
    d_threads_per_block = 256;
  }
  if((d_number_of_blobs / d_threads_per_block) < 256){
    d_threads_per_block = 128;
  }
  if((d_number_of_blobs / d_threads_per_block) < 128){
    d_threads_per_block = 128;
  }
  if((d_number_of_blobs / d_threads_per_block) < 128){
    d_threads_per_block = 64;
  }
  if((d_number_of_blobs / d_threads_per_block) < 32){
    d_threads_per_block = 128;
  }
  d_num_blocks = (d_number_of_blobs - 1) / d_threads_per_block + 1;
}

/*
  Destructor: free memory on the GPU and CPU.
*/
icc::~icc(){
  // Delete cusparse objects
  cout << "~icc STARTS " << endl;
  if(d_icc_is_initialized){
    chkErrqCusparse(cusparseDestroySolveAnalysisInfo(d_info_LT)); 
    chkErrqCusparse(cusparseDestroySolveAnalysisInfo(d_info_L)); 
    chkErrqCusparse(cusparseDestroySolveAnalysisInfo(d_info_M)); 
    hipsparseDestroyMatDescr(d_descr_L);
    hipsparseDestroyMatDescr(d_descr_M);
    chkErrqCusparse(hipsparseDestroy(d_cusp_handle));

    // Free GPU memory
    chkErrq(hipFree(d_x_gpu));
    chkErrq(hipFree(d_nnz_gpu));
    chkErrq(hipFree(d_aux_gpu));
    chkErrq(hipFree(d_cooVal_gpu));
    chkErrq(hipFree(d_cooVal_sorted_gpu));
    chkErrq(hipFree(d_cooRowInd_gpu));
    chkErrq(hipFree(d_cooColInd_gpu));
    chkErrq(hipFree(d_csrRowPtr_gpu));
  }
  cout << "~icc DONE" << endl;
}

/*
  Build sparse mobility matrix M.
*/
int icc::init_icc(){
  int N = d_number_of_blobs * 3;

  // Allocate GPU memory
  chkErrq(hipMalloc((void**)&d_x_gpu, N * sizeof(double)));
  chkErrq(hipMalloc((void**)&d_nnz_gpu, sizeof(unsigned long long int)));
  chkErrq(hipMalloc((void**)&d_aux_gpu, N * sizeof(double))); 
 
  // Copy data from CPU to GPU
  chkErrq(hipMemcpy(d_x_gpu, d_x, N * sizeof(double), hipMemcpyHostToDevice));
  d_nnz = 0;
  chkErrq(hipMemcpy(d_nnz_gpu, &d_nnz, sizeof(unsigned long long int), hipMemcpyHostToDevice));

  // Count non-zero elements in mobility matrix
  countLowerTriangularNnz<<<d_num_blocks, d_threads_per_block>>>(d_x_gpu, d_nnz_gpu, d_cutoff, d_number_of_blobs);
  // countNnz<<<d_num_blocks, d_threads_per_block>>>(d_x_gpu, d_nnz_gpu, d_cutoff, d_number_of_blobs);
  chkErrq(hipPeekAtLastError());
  chkErrq(hipMemcpy(&d_nnz, d_nnz_gpu, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
  cout << "nnz = " << d_nnz << endl;

  // Allocate GPU memory for the sparse mobility matrix
  chkErrq(hipMalloc((void**)&d_cooVal_gpu, d_nnz * sizeof(double)));
  chkErrq(hipMalloc((void**)&d_cooVal_sorted_gpu, d_nnz * sizeof(double)));
  chkErrq(hipMalloc((void**)&d_cooRowInd_gpu, d_nnz * sizeof(int)));
  chkErrq(hipMalloc((void**)&d_cooColInd_gpu, d_nnz * sizeof(int)));
  chkErrq(hipMalloc((void**)&d_csrRowPtr_gpu, ((3 * d_number_of_blobs) + 1) * sizeof(int)));

  // Build sparse mobility matrix
  d_nnz = 0;
  chkErrq(hipMemcpy(d_nnz_gpu, &d_nnz, sizeof(unsigned long long int), hipMemcpyHostToDevice));
  if(1){
    buildLowerTriangularCOOMatrix<<<d_num_blocks, d_threads_per_block>>>(d_x_gpu,
									 d_cooVal_gpu,
									 d_cooRowInd_gpu,
									 d_cooColInd_gpu,
									 d_nnz_gpu,
									 d_eta,
									 d_blob_radius,
									 d_cutoff,
									 d_number_of_blobs);
  }
  else{
    buildCOOMatrix<<<d_num_blocks, d_threads_per_block>>>(d_x_gpu,
							  d_cooVal_gpu,
							  d_cooRowInd_gpu,
							  d_cooColInd_gpu,
							  d_nnz_gpu,
							  d_eta,
							  d_blob_radius,
							  d_cutoff,
							  d_number_of_blobs);
  }
  chkErrq(hipPeekAtLastError());
  chkErrq(hipMemcpy(&d_nnz, d_nnz_gpu, sizeof(unsigned long long int), hipMemcpyDeviceToHost));
  cout << "nnz = " << d_nnz << endl;
  
  // Init cuSparse
  chkErrqCusparse(hipsparseCreate(&d_cusp_handle));
  d_base = hipsparseIndexBase_t(0);
  
  // Sort matrix to COO format
  {
    thrust::device_vector<int> vec_col(d_cooColInd_gpu, d_cooColInd_gpu + d_nnz);
    chkErrq(hipPeekAtLastError());
    thrust::device_vector<int> vec_row(d_cooRowInd_gpu, d_cooRowInd_gpu + d_nnz);
    chkErrq(hipPeekAtLastError());
    thrust::device_vector<double> vec_val(d_cooVal_gpu, d_cooVal_gpu + d_nnz);
    chkErrq(hipPeekAtLastError());
    thrust::device_vector<int> vec_col_sorted(d_nnz);
    chkErrq(hipPeekAtLastError());
    thrust::device_vector<int> vec_row_sorted(d_nnz);
    chkErrq(hipPeekAtLastError());
    thrust::device_vector<double> vec_val_sorted(d_nnz);
    chkErrq(hipPeekAtLastError());
    thrust::device_vector<unsigned long long int> vec_global_index(d_cooColInd_gpu, d_cooColInd_gpu + d_nnz);
    chkErrq(hipPeekAtLastError());
    if(0){
      cout << "Print values  ";
      thrust::copy(vec_val.begin(), vec_val.end(), std::ostream_iterator<double>(std::cout, " "));
      cout << endl;
      cout << "Print columns ";
      thrust::copy(vec_global_index.begin(), vec_global_index.end(), std::ostream_iterator<unsigned long long int>(std::cout, " "));
      cout << endl;
      cout << "Print rows    ";
      thrust::copy(vec_row.begin(), vec_row.end(), std::ostream_iterator<int>(std::cout, " "));
      cout << endl;
      // thrust::sort(d_cooRowInd, d_cooRowInd + d_nnz);
    }
    // Create global index = row*N + col
    saxpy_fast(N, vec_row, vec_global_index);
    if(0){
      // thrust::host_vector<unsigned long long int> vec_global_index_host = vec_global_index;
      cout << "Print index  ";
      thrust::copy(vec_global_index.begin(), vec_global_index.end(), std::ostream_iterator<unsigned long long int>(std::cout, " "));
      cout << endl;
    }

    // Initialize vector to [0, 1, 2, ...]
    thrust::counting_iterator<int> iter(0);
    chkErrq(hipPeekAtLastError());
    thrust::device_vector<int> indices(d_nnz);
    chkErrq(hipPeekAtLastError());
    thrust::copy(iter, iter + indices.size(), indices.begin());
    chkErrq(hipPeekAtLastError());
    // Sort the indices using the global index as the key
    thrust::sort_by_key(vec_global_index.begin(), vec_global_index.end(), indices.begin());
    chkErrq(hipPeekAtLastError());
    if(0){
      cout << "Print index  ";
      thrust::copy(vec_global_index.begin(), vec_global_index.end(), std::ostream_iterator<unsigned long long int>(std::cout, "  "));
      cout << endl;
    }

    // Sort rows, columns and values with the indices
    thrust::gather(indices.begin(), indices.end(), vec_col.begin(), vec_col_sorted.begin());
    thrust::gather(indices.begin(), indices.end(), vec_row.begin(), vec_row_sorted.begin());
    thrust::gather(indices.begin(), indices.end(), vec_val.begin(), vec_val_sorted.begin());
    if(0){
      cout << endl << endl << endl;
      cout << "Print columns ";
      thrust::copy(vec_col_sorted.begin(), vec_col_sorted.end(), std::ostream_iterator<int>(std::cout, " "));
      cout << endl;
      cout << "Print rows    ";
      thrust::copy(vec_row_sorted.begin(), vec_row_sorted.end(), std::ostream_iterator<int>(std::cout, " "));
      cout << endl;
      cout << "Print values  ";
      thrust::copy(vec_val_sorted.begin(), vec_val_sorted.end(), std::ostream_iterator<double>(std::cout, " "));
      cout << endl;
    }
    
    // Copy thrust vectors to arrays
    thrust::copy(vec_col_sorted.begin(), vec_col_sorted.end(), d_cooColInd_gpu);
    thrust::copy(vec_row_sorted.begin(), vec_row_sorted.end(), d_cooRowInd_gpu);
    thrust::copy(vec_val_sorted.begin(), vec_val_sorted.end(), d_cooVal_gpu);
  }
  // Transform sparse matrix to CSR format
  chkErrqCusparse(hipsparseXcoo2csr(d_cusp_handle, d_cooRowInd_gpu, d_nnz, N, d_csrRowPtr_gpu, d_base));
  
  // Create descriptor for matrix M
  chkErrqCusparse(hipsparseCreateMatDescr(&d_descr_M));
  // chkErrqCusparse(hipsparseSetMatType(d_descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL));
  chkErrqCusparse(hipsparseSetMatType(d_descr_M, HIPSPARSE_MATRIX_TYPE_SYMMETRIC));
  chkErrqCusparse(hipsparseSetMatIndexBase(d_descr_M, HIPSPARSE_INDEX_BASE_ZERO));
  chkErrqCusparse(hipsparseSetMatFillMode(d_descr_M, HIPSPARSE_FILL_MODE_UPPER)); // or LOWER
  chkErrqCusparse(hipsparseSetMatDiagType(d_descr_M, HIPSPARSE_DIAG_TYPE_NON_UNIT));
  // Print matrix 
  // print_csr_matrix_in_dense_format(d_cusp_handle, N, N, d_nnz, d_descr_M, d_cooVal_gpu, d_csrRowPtr_gpu, d_cooColInd_gpu, d_x);

  // Copy matrix to the CPU
  if(1){
    d_cooVal = new double [d_nnz];
    d_cooRowInd = new int [d_nnz];
    d_cooColInd = new int [d_nnz];
    d_csrRowPtr = new int [(N) + 1];
    chkErrq(hipMemcpy(d_cooVal, d_cooVal_gpu, d_nnz * sizeof(double), hipMemcpyDeviceToHost));
    chkErrq(hipMemcpy(d_cooRowInd, d_cooRowInd_gpu, d_nnz * sizeof(int), hipMemcpyDeviceToHost));
    chkErrq(hipMemcpy(d_cooColInd, d_cooColInd_gpu, d_nnz * sizeof(int), hipMemcpyDeviceToHost));
    chkErrq(hipMemcpy(d_csrRowPtr, d_csrRowPtr_gpu, ((3 * d_number_of_blobs) + 1) * sizeof(int), hipMemcpyDeviceToHost));
    
    // for(int i=0; i<d_nnz; i++){
    //   if(isnan(d_cooRowInd[i]) || isnan(d_cooColInd[i]) || isnan(d_cooVal[i])){
    // 	cout << i << " --- " << d_cooRowInd[i] << "  " << d_cooColInd[i] << "  " << d_cooVal[i] << endl;
    // 	cout << "problem in d_cooRowInd[i]) || isnan(d_cooColInd[i]) || isnan(d_cooVal[i]) " << i << " --- " << d_cooRowInd[i] << "  " << d_cooColInd[i] << "  " << d_cooVal[i] << endl;
    // 	exit(1);
    //   }
    // }
    // for(int i=0; i < ((N) + 1); i++){
    //   if(isnan(d_csrRowPtr[i])){
    // 	cout << i << " --- " << d_csrRowPtr[i] << endl;
    // 	cout << "problem in d_csrRowPtr[i] " << i << "  " << d_csrRowPtr[i] << endl;
    // 	exit(1);
    //   }
    // }
    delete[] d_cooVal;
    delete[] d_cooRowInd;
    delete[] d_cooColInd;
    delete[] d_csrRowPtr;
  }

  // Create info structure for incomplete Cholesky
  // hipsparseCreateCsric02Info(&d_info_M); for version 7.5
  cusparseCreateSolveAnalysisInfo(&d_info_M);
  hipsparseOperation_t operation = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  if(1){
    chkErrqCusparse(cusparseDcsrsv_analysis(d_cusp_handle, 
					    operation, /*HIPSPARSE_OPERATION_NON_TRANSPOSE*/
					    N,
					    d_nnz,
					    d_descr_M, 
					    d_cooVal_gpu,
					    d_csrRowPtr_gpu, 
					    d_cooColInd_gpu,
					    d_info_M));
    chkErrq(hipDeviceSynchronize());
  }

  // print_csr_matrix_in_dense_format(d_cusp_handle, N, N, d_nnz, d_descr_M, d_cooVal_gpu, d_csrRowPtr_gpu, d_cooColInd_gpu, d_x);    
  // Compute incomplete cholesky 
  if(1){
    // chkErrqCusparse(hipsparseSetMatType(d_descr_M, HIPSPARSE_MATRIX_TYPE_SYMMETRIC));
    // chkErrqCusparse(hipsparseSetMatType(d_descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL));
    chkErrqCusparse(cusparseDcsric0(d_cusp_handle,
				    operation,
				    N,
				    d_descr_M,
				    d_cooVal_gpu,
				    d_csrRowPtr_gpu,
				    d_cooColInd_gpu,
				    d_info_M));
  }
  chkErrq(hipDeviceSynchronize());

  // Print matrix 
  // print_csr_matrix_in_dense_format(d_cusp_handle, N, N, d_nnz, d_descr_M, d_cooVal_gpu, d_csrRowPtr_gpu, d_cooColInd_gpu, d_x);    

  // Create descriptor for matrix Cholesky factor L
  chkErrqCusparse(hipsparseCreateMatDescr(&d_descr_L));
  // chkErrqCusparse(hipsparseSetMatType(d_descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL));
  chkErrqCusparse(hipsparseSetMatType(d_descr_L, HIPSPARSE_MATRIX_TYPE_TRIANGULAR));
  chkErrqCusparse(hipsparseSetMatIndexBase(d_descr_L, HIPSPARSE_INDEX_BASE_ZERO));
  chkErrqCusparse(hipsparseSetMatFillMode(d_descr_L, HIPSPARSE_FILL_MODE_UPPER)); // or LOWER
  chkErrqCusparse(hipsparseSetMatDiagType(d_descr_L, HIPSPARSE_DIAG_TYPE_NON_UNIT));

  // Create info to solve with L (Cholesky factor)
  cusparseCreateSolveAnalysisInfo(&d_info_L);
  chkErrqCusparse(cusparseDcsrsv_analysis(d_cusp_handle, 
					  HIPSPARSE_OPERATION_NON_TRANSPOSE,
					  N,
					  d_nnz,
					  d_descr_L, 
					  d_cooVal_gpu,
					  d_csrRowPtr_gpu, 
					  d_cooColInd_gpu,
					  d_info_L));
  chkErrq(hipDeviceSynchronize());

  // Create info to solve with L^T
  cusparseCreateSolveAnalysisInfo(&d_info_LT);
  chkErrqCusparse(cusparseDcsrsv_analysis(d_cusp_handle, 
					  HIPSPARSE_OPERATION_TRANSPOSE,
					  N,
					  d_nnz,
					  d_descr_L, 
					  d_cooVal_gpu,
					  d_csrRowPtr_gpu, 
					  d_cooColInd_gpu,
					  d_info_LT));
  chkErrq(hipDeviceSynchronize());
  d_icc_is_initialized = 1;  
  cout << "icc_init DONE" << endl;
  return 0;
}


/*
  Muliply by Cholesky factorization L.
  L*x = b
  x_gpu and solution b_gpu are on the GPU
*/
int icc::multL_gpu(const double *x_gpu, double *b_gpu, hipsparseOperation_t operation){
  int N = d_number_of_blobs * 3;
  hipsparseMatrixType_t mat_type = hipsparseGetMatType(d_descr_L);
  double alpha = 1;
  double beta = 0;
  chkErrqCusparse(hipsparseSetMatType(d_descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL));
  chkErrqCusparse(hipsparseDcsrmv(d_cusp_handle, 
   				 operation,
   				 N,
   				 N,
   				 d_nnz,
				 &alpha,
				 d_descr_L,
				 d_cooVal_gpu,
				 d_csrRowPtr_gpu,
				 d_cooColInd_gpu,
				 x_gpu,
				 &beta,
				 b_gpu));
  chkErrq(hipDeviceSynchronize());
  chkErrqCusparse(hipsparseSetMatType(d_descr_L, mat_type));
  return 0;
}


/*
  Apply preconditioner mobility
  L^{-T} * M * L^{-1} * x = b
*/
//int icc::mult_precondM(const double *x, double *b){ 
int icc::multL(const bp::object x_obj, bp::object b_obj){ 
  // Extract pointers 
  PyObject* pobj = x_obj.ptr();
  Py_buffer pybuf;
  PyObject_GetBuffer(pobj, &pybuf, PyBUF_SIMPLE);
  void *buf = pybuf.buf;
  double *x = (double*)buf;
  PyObject* pbobj = b_obj.ptr();
  Py_buffer pybbuf;
  PyObject_GetBuffer(pbobj, &pybbuf, PyBUF_SIMPLE);
  void *bbuf = pybbuf.buf;
  double *b = (double*)bbuf;
  
  // Allocate memory
  int N = d_number_of_blobs * 3;
  double *x_gpu, *b_gpu;
  chkErrq(hipMalloc((void**)&x_gpu, N * sizeof(double)));
  chkErrq(hipMalloc((void**)&b_gpu, N * sizeof(double)));
  // Copy data from CPU to GPU
  chkErrq(hipMemcpy(x_gpu, x, N * sizeof(double), hipMemcpyHostToDevice));
  // Compute product
  hipsparseOperation_t operation = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  multL_gpu(x_gpu, b_gpu, operation);
  // Copy data from GPU to CPU
  chkErrq(hipMemcpy(b, b_gpu, N * sizeof(double), hipMemcpyDeviceToHost));
  // Free memory
  chkErrq(hipFree(b_gpu));
  chkErrq(hipFree(x_gpu));
  return 0;
}


/*
  Solve with Cholesky factor L
  L*x = b
  solution x_gpu and RHS b_gpu are in the GPU
*/
int icc::solveL_gpu(const double *b_gpu, double *x_gpu){ 
  int N = d_number_of_blobs * 3;
  double alpha = 1;
  alpha = 1;
  chkErrqCusparse(cusparseDcsrsv_solve(d_cusp_handle,
				       HIPSPARSE_OPERATION_NON_TRANSPOSE,
				       N, 
				       &alpha,
				       d_descr_L,
				       d_cooVal_gpu,
				       d_csrRowPtr_gpu, 
				       d_cooColInd_gpu,
				       d_info_L,
				       b_gpu, 
				       x_gpu));
  chkErrq(hipDeviceSynchronize());
  return 0;
}

/*
  Solve with Cholesky (transpose) factor L^T
  L^T*x = b
  solution x_gpu and RHS b_gpu are on the GPU
*/
int icc::solveLT_gpu(const double *b_gpu, double *x_gpu){
  int N = d_number_of_blobs * 3;
  double alpha = 1;
  alpha = 1;
  chkErrqCusparse(cusparseDcsrsv_solve(d_cusp_handle,
				       HIPSPARSE_OPERATION_TRANSPOSE,
				       N, 
				       &alpha,
				       d_descr_L,
				       d_cooVal_gpu,
				       d_csrRowPtr_gpu, 
				       d_cooColInd_gpu,
				       d_info_LT,
				       b_gpu, 
				       x_gpu));
  chkErrq(hipDeviceSynchronize());
  return 0;
}


/*
  Apply preconditioner mobility
  L^{-T} * M * L^{-1} * x = b
*/
int icc::mult_precondM_gpu(const double *x_gpu, double *b_gpu){
  // First, solve L*b=x
  solveL_gpu(x_gpu, d_aux_gpu);
  // Second, apply mobility M*x = b
  velocity_from_force<<<d_num_blocks, d_threads_per_block>>>(d_x_gpu,
							     d_aux_gpu,					
							     b_gpu,
							     d_number_of_blobs,
							     d_eta,
							     d_blob_radius);
  chkErrq(hipDeviceSynchronize());
  chkErrq(hipMemcpy(d_aux_gpu, b_gpu, 3 * d_number_of_blobs * sizeof(double), hipMemcpyDeviceToDevice)); 
  // Third, solve L.T*b = x
  solveLT_gpu(d_aux_gpu, b_gpu);
  return 0;
}


/*
  Apply preconditioner mobility
  L^{-T} * M * L^{-1} * x = b
*/
//int icc::mult_precondM(const double *x, double *b){ 
int icc::mult_precondM(const bp::object x_obj, bp::object b_obj){ 
  // Extract pointers 
  PyObject* pobj = x_obj.ptr();
  Py_buffer pybuf;
  PyObject_GetBuffer(pobj, &pybuf, PyBUF_SIMPLE);
  void *buf = pybuf.buf;
  double *x = (double*)buf;
  PyObject* pbobj = b_obj.ptr();
  Py_buffer pybbuf;
  PyObject_GetBuffer(pbobj, &pybbuf, PyBUF_SIMPLE);
  void *bbuf = pybbuf.buf;
  double *b = (double*)bbuf;
  
  // Allocate memory
  int N = d_number_of_blobs * 3;
  double *x_gpu, *b_gpu;
  chkErrq(hipMalloc((void**)&x_gpu, N * sizeof(double)));
  chkErrq(hipMalloc((void**)&b_gpu, N * sizeof(double)));
  // Copy data from CPU to GPU
  chkErrq(hipMemcpy(x_gpu, x, N * sizeof(double), hipMemcpyHostToDevice));
  // Compute product
  mult_precondM_gpu(x_gpu, b_gpu);
  // Copy data from GPU to CPU
  chkErrq(hipMemcpy(b, b_gpu, N * sizeof(double), hipMemcpyDeviceToHost));
  // Free memory
  chkErrq(hipFree(b_gpu));
  chkErrq(hipFree(x_gpu));
  return 0;
}


int main(){
  // Define parameters
  int status;
  double aux;
  double blob_radius = 1.0;
  double eta = 1.0;
  double cutoff = 75;
  int number_of_blobs = 1000;
  int N = number_of_blobs * 3;

  // Create CPU arrays
  double *x; 
  if(0){
    x = new double [N];
    for(int i=0; i<(N); i++){
      x[i] = 1000.0 * rand() / RAND_MAX;
    }
  }
  else{
    ifstream coor("shell_3d_Nblob_1_R_h_1.clones");
    coor >> number_of_blobs;
    N = number_of_blobs * 3;
    x = new double [N];
    for(int i=0; i<number_of_blobs; i++){
      coor >> x[i*3] >> x[i*3+1] >> x[i*3+2];
      coor >> aux >> aux >> aux >> aux;
    }
    coor.close();
  }
 
  // Create icc object
  icc icc_obj = icc(blob_radius, eta, cutoff, number_of_blobs, x);
  
  // Build sparse mobility matrix
  status = icc_obj.init_icc();
  cout << "Build sparse mobility matrix = " << status << endl;
  
  // Test solve L*x = b
  double *b = new double[N];
  for(int i=0; i<N; i++){
    b[i] = 1.0;
  }
  double *b_gpu, *x_gpu;
  chkErrq(hipMalloc((void**)&b_gpu, N * sizeof(double)));  
  chkErrq(hipMalloc((void**)&x_gpu, N * sizeof(double)));  
  // Move info to gpu
  chkErrq(hipMemcpy(x_gpu, b, N * sizeof(double), hipMemcpyHostToDevice));    
  chkErrq(hipMemcpy(b_gpu, b, N * sizeof(double), hipMemcpyHostToDevice));      

  // Compute RHS = L*x
  hipsparseOperation_t operation = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  icc_obj.multL_gpu(x_gpu, b_gpu, operation);
  chkErrq(hipMemcpy(b, b_gpu, N * sizeof(double), hipMemcpyDeviceToHost));    
  for(int i=0; i<N; i++){
    if(isnan(b[i])){
      cout << "icc_obj.multL_gpu -- i, b = " << i << "    " << b[i] << endl;
      exit(1);
    }
  }
  // Solve system L*x = RHS
  icc_obj.solveL_gpu(b_gpu, x_gpu);
  chkErrq(hipMemcpy(x, x_gpu, N * sizeof(double), hipMemcpyDeviceToHost));    
  chkErrq(hipDeviceSynchronize());
  for(int i=0; i<N; i++){
    cout << "icc_obj.solveL_gpu i, x = " << i << "    " << x[i] << endl;
    b[i] = 1.0;
  }
  chkErrq(hipMemcpy(x_gpu, b, N * sizeof(double), hipMemcpyHostToDevice));    

  cout << endl;
  if(1){
    // Compute RHS = L.T*x
    operation = HIPSPARSE_OPERATION_TRANSPOSE;
    icc_obj.multL_gpu(x_gpu, b_gpu, operation);
    chkErrq(hipMemcpy(b, b_gpu, N * sizeof(double), hipMemcpyDeviceToHost));    
    for(int i=0; i<N; i++){
      cout << "icc_obj.multL_gpu TRANSPOSE i, b = " << i << "    " << b[i] << endl;
    }
    // Solve system L*x = RHS
    icc_obj.solveLT_gpu(b_gpu, x_gpu);
    chkErrq(hipMemcpy(x, x_gpu, N * sizeof(double), hipMemcpyDeviceToHost));    
    chkErrq(hipDeviceSynchronize());
    for(int i=0; i<N; i++){
      cout << "icc_obj.solveLT_gpu TRANSPOSE i, x = " << i << "    " << x[i] << endl;
      b[i] = 1.0;
    }
    chkErrq(hipMemcpy(x_gpu, b, N * sizeof(double), hipMemcpyHostToDevice));    
  }
  cout << endl;

  if(1){
    icc_obj.mult_precondM_gpu(x_gpu, b_gpu);
    chkErrq(hipMemcpy(b, b_gpu, N * sizeof(double), hipMemcpyDeviceToHost));    
    for(int i=0; i<N; i++){
      cout << "icc_obj.mult_precondM_gpu i, b = " << i << "    " << b[i] << endl;
    }
  }

  


  // Free GPU memory
  chkErrq(hipFree(x_gpu));
  chkErrq(hipFree(b_gpu));
  // Free CPU memory
  delete[] b;
  cout << "before x" << endl;
  delete[] x;
  cout << "# End" << endl;
  return 0;
}




BOOST_PYTHON_MODULE(icc_ext)
{
  using namespace boost::python;
  boost::python::numeric::array::set_module_and_type("numpy", "ndarray");
  class_<icc>("icc", init<const double, const double, const double, const int, const double*>())
    .def(init<const double, const double, const double, const int, bp::object>())
    .def("init_icc", &icc::init_icc)
    .def("mult_precondM_gpu", &icc::mult_precondM_gpu)
    .def("mult_precondM", &icc::mult_precondM)
    .def("multL", &icc::multL)
    ;
}
